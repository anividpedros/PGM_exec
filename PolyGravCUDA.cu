#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <fstream>
#include "Polygon.hpp"
#include "Matrix.hpp"
#include "Vect.hpp"
#include "book.h"

const int blocksPerGrid = 32;
const int threadsPerBlock = 256;


__global__ void FaceTerm(int* NOF, double* ListTri, double* F, double* X, double* Y, double* Z, double* Xsc, double* acc)
{

__shared__ double ax[threadsPerBlock];
__shared__ double ay[threadsPerBlock];
__shared__ double az[threadsPerBlock];

int i = threadIdx.x + blockIdx.x*blockDim.x;
int cacheIndex = threadIdx.x;

int v1, v2, v3;

double r1x, r1y, r1z;
double r2x, r2y, r2z;
double r3x, r3y, r3z;

double R1, R2, R3;

double dot_num, dot_23, dot_31, dot_12;

double wf;

double tempx = 0.0;
double tempy = 0.0;
double tempz = 0.0;

while(i < *NOF)
{

	v1 = (int) ListTri[3*i];
	v2 = (int) ListTri[3*i + 1];
	v3 = (int) ListTri[3*i + 2];

	//Define First Vertex
	r1x = X[v1-1] - Xsc[0];
	r1y = Y[v1-1] - Xsc[1];
	r1z = Z[v1-1] - Xsc[2];
	R1  = sqrt(r1x*r1x + r1y*r1y + r1z*r1z);

	//Define Second Vertex
	r2x = X[v2-1] - Xsc[0];
	r2y = Y[v2-1] - Xsc[1];
	r2z = Z[v2-1] - Xsc[2];
	R2  = sqrt(r2x*r2x + r2y*r2y + r2z*r2z);

	//Define Third Vertex
	r3x = X[v3-1] - Xsc[0];
	r3y = Y[v3-1] - Xsc[1];
	r3z = Z[v3-1] - Xsc[2];
	R3  = sqrt(r3x*r3x + r3y*r3y + r3z*r3z);


	dot_num = r1x*(r2y*r3z - r2z*r3y) + r1y*(r2z*r3x - r2x*r3z) + r1z*(r2x*r3y - r2y*r3x);

	dot_23 = r2x*r3x + r2y*r3y + r2z*r3z;
	dot_31 = r3x*r1x + r3y*r1y + r3z*r1z;
	dot_12 = r1x*r2x + r1y*r2y + r1z*r2z;

	//Define w_f
	wf = 2*atan2(dot_num, R1*R2*R3 + R1*dot_23 + R2*dot_31 + R3*dot_12);

	//Store acceleration
	tempx += wf*(F[9*i + 0]*r1x + F[9*i + 1]*r1y + F[9*i + 2]*r1z);
	tempy += wf*(F[9*i + 3]*r1x + F[9*i + 4]*r1y + F[9*i + 5]*r1z);
	tempz += wf*(F[9*i + 6]*r1x + F[9*i + 7]*r1y + F[9*i + 8]*r1z);

	i += blockDim.x * gridDim.x;
}

ax[cacheIndex] = tempx;
ay[cacheIndex] = tempy;
az[cacheIndex] = tempz;

// synchronize threads in the block
__syncthreads();


int j = blockDim.x/2;

while(j != 0){

	if(cacheIndex < j)
	{
		ax[cacheIndex] += ax[cacheIndex + j];
		ay[cacheIndex] += ay[cacheIndex + j];
		az[cacheIndex] += az[cacheIndex + j];
	}
	__syncthreads();

	j /= 2;
}

if(cacheIndex == 0)
{
	acc[3*blockIdx.x + 0] = ax[0];
	acc[3*blockIdx.x + 1] = ay[0];
	acc[3*blockIdx.x + 2] = az[0];
}


}



__global__ void EdgeTerm(int* NOE, double* ListE, double* E, double* X, double* Y, double* Z, double* Xsc, double* acc)
{

__shared__ double ax[threadsPerBlock];
__shared__ double ay[threadsPerBlock];
__shared__ double az[threadsPerBlock];

	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int cacheIdx = threadIdx.x;

	int e1, e2;

	double r1x, r1y, r1z, R1;
	double r2x, r2y, r2z, R2;

	double dex, dey, dez, de;

	double Le;

	double tempx = 0.0;
	double tempy = 0.0;
	double tempz = 0.0;

	while(i < *NOE)
	{

		e1 = (int) ListE[2*i];
		e2 = (int) ListE[2*i + 1];

		//Define First Vertex
		r1x = X[e1-1] - Xsc[0];
		r1y = Y[e1-1] - Xsc[1];
		r1z = Z[e1-1] - Xsc[2];
		R1  = sqrt(r1x*r1x + r1y*r1y + r1z*r1z);

		//Define Second Vertex
		r2x = X[e2-1] - Xsc[0];
		r2y = Y[e2-1] - Xsc[1];
		r2z = Z[e2-1] - Xsc[2];
		R2  = sqrt(r2x*r2x + r2y*r2y + r2z*r2z);

		dex = r2x - r1x;
		dey = r2y - r1y;
		dez = r2z - r1z;
		de = sqrt(dex*dex + dey*dey + dez*dez);

		Le  = log((R1 + R2 + de)/(R1 + R2 - de));

		tempx += Le*(E[9*i + 0]*r1x + E[9*i + 1]*r1y + E[9*i + 2]*r1z);
		tempy += Le*(E[9*i + 3]*r1x + E[9*i + 4]*r1y + E[9*i + 5]*r1z);
		tempz += Le*(E[9*i + 6]*r1x + E[9*i + 7]*r1y + E[9*i + 8]*r1z);

		i += blockDim.x * gridDim.x;

	}

	ax[cacheIdx] = tempx;
	ay[cacheIdx] = tempy;
	az[cacheIdx] = tempz;

	__syncthreads();


	int j = blockDim.x/2;

	while(j!=0)
	{
	if(cacheIdx < j)
	{
	 ax[cacheIdx] += ax[cacheIdx + j];
	 ay[cacheIdx] += ay[cacheIdx + j];
	 az[cacheIdx] += az[cacheIdx + j];
	}

	__syncthreads();
	j /= 2;

	}

	if(cacheIdx == 0){
	 acc[3*blockIdx.x + 0] = ax[0];
	 acc[3*blockIdx.x + 1] = ay[0];
	 acc[3*blockIdx.x + 2] = az[0];
	} 

}



int main()
{

std::ifstream GRAV ("GravityFile.txt");

Polygon CG67P(GRAV);

int NOV = CG67P.GetNOV();

// Vertex Coordinates
Vect X = CG67P.GetX();
Vect Y = CG67P.GetY();
Vect Z = CG67P.GetZ();

// Load Vertex Coordinates on GPU
double* d_X;
double* d_Y;
double* d_Z;

HANDLE_ERROR(hipMalloc((void**)&d_X, NOV*sizeof(double)));
HANDLE_ERROR(hipMalloc((void**)&d_Y, NOV*sizeof(double)));
HANDLE_ERROR(hipMalloc((void**)&d_Z, NOV*sizeof(double)));

HANDLE_ERROR(hipMemcpy(d_X, &X[0], NOV*sizeof(double), hipMemcpyHostToDevice));
HANDLE_ERROR(hipMemcpy(d_Y, &Y[0], NOV*sizeof(double), hipMemcpyHostToDevice));
HANDLE_ERROR(hipMemcpy(d_Z, &Z[0], NOV*sizeof(double), hipMemcpyHostToDevice));


// Facet Term
int NOF = CG67P.GetNOF();


Vect ListTri = CG67P.GetListTri();
//Matrix ListN   = KW4A.GetListN();
Vect F       = CG67P.GetF();


//Copy Variables on GPU
int*    d_NOF;
double* d_ListTri;
//double* d_ListN;
double* d_F;


HANDLE_ERROR(hipMalloc((void**) &d_NOF, sizeof(int)));
HANDLE_ERROR(hipMalloc((void**) &d_ListTri, NOF*3*sizeof(double)));
//hipMalloc((void**) &d_ListN, NOF*3*sizeof(double));
HANDLE_ERROR(hipMalloc((void**) &d_F, NOF*9*sizeof(double)));

HANDLE_ERROR(hipMemcpy(d_NOF, &NOF, sizeof(int), hipMemcpyHostToDevice));
HANDLE_ERROR(hipMemcpy(d_ListTri, &ListTri[0], NOF*3*sizeof(double), hipMemcpyHostToDevice));
//hipMemcpy(d_ListN, &ListN[0], NOF*3*sizeof(double), hipMemcpyHostToDevice);
HANDLE_ERROR(hipMemcpy(d_F, &F[0], NOF*9*sizeof(double), hipMemcpyHostToDevice));


//Define Spacecraft position vector
Vect Xsc(3);

Xsc[0] = 1000.0;
Xsc[1] = 0.;
Xsc[2] = 0.;


double* d_Xsc;

HANDLE_ERROR(hipMalloc((void**) &d_Xsc, 3*sizeof(double)));

HANDLE_ERROR(hipMemcpy(d_Xsc, &Xsc[0], 3*sizeof(double), hipMemcpyHostToDevice));

double* d_aF;

HANDLE_ERROR(hipMalloc((void**) &d_aF, blocksPerGrid*3*sizeof(double)));

FaceTerm<<<blocksPerGrid,threadsPerBlock>>>(d_NOF, d_ListTri, d_F, d_X, d_Y, d_Z, d_Xsc, d_aF);

Vect accF(blocksPerGrid*3);

HANDLE_ERROR(hipMemcpy(&accF[0], d_aF, blocksPerGrid*3*sizeof(double), hipMemcpyDeviceToHost));



// Edge Term
int  NOE    = CG67P.GetNOE();
Vect ListE  = CG67P.GetListE();
Vect E      = CG67P.GetE();

int* d_NOE;
double* d_ListE;
double* d_E;

HANDLE_ERROR(hipMalloc((void**) &d_NOE, sizeof(int)));
HANDLE_ERROR(hipMalloc((void**) &d_ListE, NOE*2*sizeof(double)));
HANDLE_ERROR(hipMalloc((void**) &d_E, NOE*9*sizeof(double)));

HANDLE_ERROR(hipMemcpy(d_NOE, &NOE, sizeof(int), hipMemcpyHostToDevice));
HANDLE_ERROR(hipMemcpy(d_ListE, &ListE[0], NOE*2*sizeof(double), hipMemcpyHostToDevice));
HANDLE_ERROR(hipMemcpy(d_E, &E[0], NOE*9*sizeof(double), hipMemcpyHostToDevice));


double* d_aE;

HANDLE_ERROR(hipMalloc((void**) &d_aE, blocksPerGrid*3*sizeof(double)));

EdgeTerm<<<blocksPerGrid,threadsPerBlock>>>(d_NOE, d_ListE, d_E, d_X, d_Y, d_Z, d_Xsc, d_aE);

Vect accE(blocksPerGrid*3);

HANDLE_ERROR(hipMemcpy(&accE[0], d_aE, blocksPerGrid*3*sizeof(double),hipMemcpyDeviceToHost));

double ax = 0.0;
double ay = 0.0;
double az = 0.0;

for(int i = 0; i < blocksPerGrid; i++)
{
ax += accF[3*i]   - accE[3*i];
ay += accF[3*i+1] - accE[3*i+1];
az += accF[3*i+2] - accE[3*i+2];
}


double Gs = CG67P.GetGs();

//std::cout << "Gs: " << Gs << std::endl;

Vect acc(3);
acc[0] = Gs*ax;
acc[1] = Gs*ay;
acc[2] = Gs*az;


std::cout << "Final Acceleration:" << std::endl; 
disp(acc);



// Free CUDA MEMORY
//Spacecraft Coordinates
hipFree(d_Xsc);

//Vertex Coordinates
hipFree(d_X);
hipFree(d_Y);
hipFree(d_Z);

//Facet Term
hipFree(d_NOF);
hipFree(d_ListTri);
//hipFree(d_ListN);
hipFree(d_F);
hipFree(d_aF);

//Edge Term
hipFree(d_NOE);
hipFree(d_ListE);
hipFree(d_E);
hipFree(d_aE);

return 0;

}
